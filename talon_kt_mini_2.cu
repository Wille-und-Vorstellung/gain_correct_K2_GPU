#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "ReadFile.h"
#include "ReadFile.c"
#define LINE 1024
#define NAME 1024
#include "time.h"
#include "omp.h"
#include "hip/hip_runtime.h"

#define GRID_BLOCK 1
#define BLOCK_SIZE 1
#define GTHREAD_N ( GRID_BLOCK * BLOCK_SIZE )
#define GIGA 1073741824
#define MP_THREAD 10

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads);
int dispatcher_gpu(float *coord_l, float *gain_l , long size_x, long size_y, long slice_n, int type);

__global__ void mutiplier_kernel_type_c_exp(float *coord, float *gain, long long, long long, long long);
__global__ void mutiplier_kernel_type_c_exp_mini_2(float *coord, float *gain, long long, long long, long long);
void mutiplier_kernel_type_c_exp_test(float *coord, float *gain, long long total_s, long long single_s, long long unit_n);


int main(int argc, char *argv[])
{

	//printf("raw_name %s, out_name %s, gain_name %s\n",raw_name,out_name,gain_name);
	char *raw_name = NULL, *gain_name = NULL, *out_name = NULL;
	MrcHeader *inhead=NULL;
    defect_gain_correct(raw_name,gain_name,out_name,inhead, MP_THREAD );
	/*
	free(raw_name);
	free(out_name);
	free(gain_name);
	*/
	return 0;
}

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads){
	
	//revise input image header then written for output file
	int size_x=5000;
	int size_y=2000;
	int slice_n=2;
    int indicator=0;
	int result=0;
	int sum=0;
	float *coor_xy = NULL;
	float *gain_xy = NULL;

	coor_xy = (float *)malloc( sizeof(int)*size_x*size_y*slice_n );
	gain_xy = (float *)malloc( sizeof(int)*size_x*size_y );

	for (int j=0; j < size_x*size_y; j++ ){
		gain_xy[j] = 2;
	}
	for (int k=0; k < size_x*size_y*slice_n; k++ ){
		if ( k < size_x*size_y  ) {
			coor_xy[k] = 1;
		}
		else {
			coor_xy[k] = 2;
		}
	}

	indicator=dispatcher_gpu( coor_xy, gain_xy, size_x, size_y, slice_n, 0 );

	for(int i=0; i<size_x*size_y*slice_n; i++){
		if ( i < size_x*size_y && coor_xy[i] != 2 ) {
			result+=1;
		}
		else if( i > size_x*size_y && coor_xy[i] != 4 ) {
			result+=1;
		}
		else;
		sum += coor_xy[i];
	}

	printf("result: %ld, %d\n", result, sum );
	return 0;
}

/**********************/

int dispatcher_gpu(float *coord_l, float *gain_l , long size_x, long size_y, long slice_n, int type){
	//set up cuda 
	hipSetDevice(0);	
	void *device_coord=NULL; //potential: change all to double
	void *device_gain=NULL;
	//long *d_x=NULL, *d_y=NULL,*d_n=NULL;
	//void *temp_coord=NULL;
	//void *temp_gain=NULL;
	hipError_t f1, f2;

	long long total_s=0, single_s=0, unit_n=0;
	single_s = size_x * size_y;
	total_s = single_s * slice_n;

	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);

	printf("Initialize allocation.\n");
	printf("Required total(GB): %ld \n", (sizeof(float)*total_s + sizeof(float)*single_s)/GIGA);
	printf("Required each: %ld -- %ld \n", sizeof(float)*total_s, sizeof(float)*single_s);
	f1 = hipMalloc( (void **)&device_coord,  sizeof(float)*total_s  );
	f2 = hipMalloc( (void **)&device_gain,  sizeof(float)*single_s );
	//f3 = hipMalloc( (void **)&device_src, sizeof(char*)*src_size );
	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("cuda memory allocation failed: %s -- %s \n", f1, f2);
		exit(4);
	}
	
	/*
	hipMalloc( (void **)&device_coord,  sizeof(float)*size_x*size_y*slice_n  );
	hipMalloc( (void **)&device_gain,  sizeof(float)*size_x*size_y );
	hipMalloc( (void **)&device_src, sizeof(char*)*src_size );
	*/
	printf("Allocation done.\n");
	printf("status: %s -- %s \n", f1, f2);

	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);
	/*
	hipMalloc( (void **)&d_x, sizeof(long) );
	hipMalloc( (void **)&d_y, sizeof(long) );
	hipMalloc( (void **)&d_n, sizeof(long) );
	*/
	//data transfer to device
	printf("Initialize Memcpy: host->device\n");
	
	///* Kernel validation
	f1 = hipMemcpy( device_coord, coord_l, sizeof(float)*total_s, hipMemcpyHostToDevice );
	f2 = hipMemcpy( device_gain, gain_l, sizeof(float)*single_s, hipMemcpyHostToDevice );
	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("hipMemcpy failed(H->D): %s -- %s \n", f1, f2);
		exit(5);
	}
	//*/
	printf("Memcpy Done: host->device\n");

	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);

	/*
	hipMemcpy( d_x, &size_x, sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy( d_y, &size_y, sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy( d_n, &slice_n, sizeof(long), hipMemcpyHostToDevice);
	*/
	//activate kernel
	unit_n = (total_s/( GTHREAD_N )) + 1; 
	switch(type){
		case 0://c
			printf("waypoint c\n");
			//mutiplier_kernel_type_c_exp<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, total_s, single_s, unit_n );
			mutiplier_kernel_type_c_exp_mini_2<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, total_s, single_s, unit_n );
			/*
			unit_n = total_s;
			mutiplier_kernel_type_c_exp<<< 1, 1 >>>( (float*)device_coord, (float*)device_gain, total_s, single_s, unit_n );
			*/
			////Kernel validation
			/*
			unit_n = total_s;
			mutiplier_kernel_type_c_exp_test( coord_l, gain_l, total_s, single_s, unit_n);
			*/
			printf("C out\n");
			break;
		case 1://s
			//mutiplier_kernel_type_su<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, device_src, size_x, size_y, slice_n );			
			break;
		case 2://f
			//mutiplier_kernel_type_f<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, device_src, size_x, size_y, slice_n );
			break;
		case 6://u
			//mutiplier_kernel_type_su<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, device_src, size_x, size_y, slice_n );			
			break;
		default:
			printf("Well...Houston, we've got some problem...\n");
			exit(1);
	}
	//thread synchronization 
	f1 = hipDeviceSynchronize();
	if ( f1 != hipSuccess ){
		printf("cuda sync failed: %s \n", f1 );
		exit(9);
	}
	printf("Sync Done\n");

	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);

	//data transfer from device
	///* Kernel validation
	printf("Initialize Memcpy: device->host\n");
	
	f1 = hipMemcpy( coord_l, device_coord, sizeof(float)*total_s, hipMemcpyDeviceToHost );

	//GRAM allocation check
	if ( f1 != hipSuccess ){
		printf("hipMemcpy failed(D->H): %s \n", f1 );
		exit(6);
	}
	printf("Memcpy Done: device->host\n");

	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);
	//*/
	//hipMemcpy( coord_l, device_coord, size_x*size_y*slice_n, hipMemcpyDeviceToHost );
	//hipMemcpy( gain_l, device_gain, size_x*size_y,hipMemcpyDeviceToHost );
	
	//clean up
	printf("Initialize hipFree\n");
	f1 = hipFree( device_coord );
	f2 = hipFree( device_gain );
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("hipFree failed: %s -- %s \n", f1, f2);
		exit(7);
	}
	printf("hipFree Done\n");
	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);

	//hipFree( device_src );
	/*
	hipFree( d_x );
	hipFree( d_y );
	hipFree( d_n );
	*/
	printf("Dispatcher out\n");
	return 117;
}

__global__ void mutiplier_kernel_type_c_exp(float *coord, float *gain, long long total_s, long long single_s, long long unit_n){
	//execute all those mutiplications of type_c data
	long long index=0;
	long long i = 0; 
	//long long unit_n = (size_x*size_y*slice_n/( GTHREAD_N ))+1;
	//long long unit_n = unit_s;
	//long unit_n = UNIT_N;
	//long unit_n = (size_x*size_y*slice_n);
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long long slice_c=0;

	printf("index check: %d\n ", index);
	//printf("Size check: %d \n", size_x*size_y*slice_n);
	printf("Size check: %d \n", total_s);
	printf("unit_n check: %d \n", unit_n);
	printf("input check: %p -- %p \n", coord, gain);

	for (i=0; i<unit_n; i++){
		//printf("head-> %d\n", i);
		if (index*unit_n + i >= total_s ){ //boundary check
			printf("X: %d\n", index*unit_n + i); //POTENTIAL: return a counter through pointer
			return;
		}
		//printf("progress: %d \n", i);
		//slice_c = (index*unit_n+i)%(size_x*size_y);
		slice_c = (index*unit_n+i)%(single_s);
		coord[index*unit_n+i] = (coord[index*unit_n+i] * gain[slice_c]);
		//printf("tail-> %d\n", i);
	}
	//__threadfence()
	printf("Kernel out: %d\n", i);
	return;
}

__global__ void mutiplier_kernel_type_c_exp_mini_2(float *coord, float *gain, long long total_s, long long single_s, long long unit_n){
	//execute all those mutiplications of type_c data
	long long index=0;
	long long i = 0; 
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long long slice_c=0;
	
	long long index_1=0;

	printf("index check: %d\n ", index);
	printf("Size check: %d \n", total_s);
	printf("unit_n check: %d \n", unit_n);
	printf("input check: %p -- %p \n", coord, gain);

	index_1 = index*unit_n;
	printf("Local iteration start\n");
	for (i=0; i<unit_n; i++){
		
		if (index_1 >= total_s ){ //boundary check
			printf("Y: %d\n", index_1); //POTENTIAL: return a counter through pointer
			return;
		}
		
		slice_c = (index_1)%(single_s);
		//coord[index_1] = (coord[index_1] * gain[slice_c]);
		index_1+=1;
	}
	
	printf("Kernel_mini out: %d\n", i);
	return;
}

void mutiplier_kernel_type_c_exp_test(float *coord, float *gain, long long total_s, long long single_s, long long unit_n){
	//execute all those mutiplications of type_c data
	long long index=0; 
	//long long unit_n = (size_x*size_y*slice_n/( GTHREAD_N ))+1;
	//long long unit_n = unit_s;
	//long unit_n = UNIT_N;
	//long unit_n = (size_x*size_y*slice_n);
	//index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	printf("index check: %d\n ", index);
	//printf("Size check: %d \n", size_x*size_y*slice_n);
	printf("Size check: %d \n", total_s);
	printf("unit_n check: %d \n", unit_n);

	for (int i=0; i<unit_n; i++){
		//printf("head-> %d\n", i);
		if (index*unit_n + i >= total_s ){ //boundary check
			printf("X: %d", index*unit_n + i);
			return;
		}
		//printf("progress: %d \n", i);
		//slice_c = (index*unit_n+i)%(size_x*size_y);
		slice_c = (index*unit_n+i)%(single_s);
		coord[index*unit_n+i] = (coord[index*unit_n+i] * gain[slice_c]);
		//printf("tail-> %d\n", i);
	}
	//__threadfence()
	printf("Kernel out\n");
	return;
}

