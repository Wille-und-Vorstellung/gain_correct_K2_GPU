#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "ReadFile.h"
#include "ReadFile.c"
#define LINE 1024
#define NAME 1024
#include "time.h"
#include "omp.h"
#include "hip/hip_runtime.h"

//2 GPU, "transfer kernel" remained, old version

#define GRID_BLOCK 16 //16
#define BLOCK_SIZE 32 //32
#define GTHREAD_N ( GRID_BLOCK * BLOCK_SIZE )
#define GIGA 1073741824
#define MP_THREAD 10
#define TEST_RUN false

#define SIZE 2000 //debug only, alpha_test()
#define B_SIZE 28125 // debug only, beta_test()
#define B_GAIN_S 5625

#define GPU_DEVICE_0 0 
#define GPU_DEVICE_1 1

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads);
__global__ void mutiplier_kernel_syndicate(float *coord, float *gain, long long, long long, long long, long long);
__global__ void transfer_kernel( float *coord, char *src, long long total_s, long long unit_n, int step );
float dispatcher_gpu_syndicate(float *coord_l, void *src, float *gain_l , long size_x, long size_y, long slice_n, int type);
bool cudaErrorCheck( hipError_t, int );

bool alpha_test(); //type 2 input
bool beta_test( int type = 0 ); //type 0, 1, 6

int main(int argc, char *argv[]){
	clock_t global_start_ts = 0, global_end_ts = 0; // ts -> time stamp 
	global_start_ts = clock();

	bool k = true, h = true;
	if ( TEST_RUN ){
		//k = alpha_test();
		h = beta_test();
		if ( k == false ){
			printf("Alpha Failed \n");
			return 1;
		}
		if ( h == false ){
			printf( "Beta Failed \n" );
			return 2;
		}
		printf( "Passed \n" );
		return 0;
	}

    char *raw_name,*out_name,*gain_name;
    FILE *file;
    MrcHeader *inhead;

    raw_name=(char *)malloc(NAME*sizeof(char));
    out_name=(char *)malloc(NAME*sizeof(char));
    gain_name=(char *)malloc(NAME*sizeof(char));

    inhead=(MrcHeader *)malloc(sizeof(MrcHeader));

    if(argc!=4)
    {
        printf("Please input: raw_image gain_name out_name threads\n");
        return 1;
    }
    raw_name=argv[1];
    gain_name=argv[2];
    out_name=argv[3];

    file=fopen(raw_name,"rb");	
    mrc_read_head(file,inhead);
    fclose(file);

    defect_gain_correct(raw_name,gain_name,out_name,inhead, MP_THREAD );
	global_end_ts = clock();

	printf( "Total time cost: %ds \n", (global_end_ts - global_start_ts)/CLOCKS_PER_SEC );
	return 0;
}

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads){
	float *gain_xy=NULL;
	float *coor_xy=NULL;
	int size_bit;
	long int n;
	FILE *input,*output,*gain_f;
	char *input_byte_c=NULL;short *input_byte_s=NULL;float *input_byte_f=NULL;short *input_byte_u=NULL;
	srand((unsigned) time(NULL));
	void *source =NULL;

	//open input output and gain file
	printf("Start defect and gain correction|\ninput %s output %s gain %s\n",fin,fout,gain);
	input=fopen(fin,"rb");
	gain_f=fopen(gain,"rb");

	//revise input image header then written for output file
	int size_x=head->nx;
	int size_y=head->ny;
	int slice_n=head->nz;
    int file_type=head->mode;
	head->mode=2;
    switch(file_type){
        case 0:	size_bit=1;
				break;
        case 1:	size_bit=2;
				break;
        case 2:	size_bit=4;
				break;
        case 6:	size_bit=2;
				break;
        default:
				printf("File type error!");
    }

	//calculate the size of the input and gain file by byte.
	fseek(input,0,SEEK_END);
	long int input_size=ftell(input);
	rewind(input);

	fseek(gain_f,0,SEEK_END);
	long int gain_f_size=ftell(gain_f);
	rewind(gain_f);

	//skip header
	fseek(input,1024,0);
	fseek(gain_f,1024,0);

	//malloc memory for the file pointer
	float *gain_byte=(float*)malloc(sizeof(char*)*(gain_f_size-1024));

	printf("pointer check(pre-allocate): %p -- %p \n", coor_xy, gain_xy);
	gain_xy=(float*)malloc(sizeof(float)*size_y*size_x);
	coor_xy=(float*)malloc(sizeof(float)*size_y*size_x*slice_n);
	printf("pointer check(post-allocate): %p -- %p \n", coor_xy, gain_xy);

	//read the input and gain file into memory
	//note: the byte size(second parameter) is basen on the type of the file
	printf( "--> Input Mrc type: %d \n", file_type );
	printf("Before dispatch, statues: src_size - %d, size_x - %d, size_y - %d, slice_n %d \n", input_size-1024, size_x, size_y, slice_n );

	switch(file_type){
        case 0:	size_bit=1;input_byte_c=(char*)malloc(sizeof(char)*(input_size-1024));
				fread(input_byte_c,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_c;
				break;
        case 1:	size_bit=2;input_byte_s=(short*)malloc(sizeof(char)*(input_size-1024));			fread(input_byte_s,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_s;
				break;
        case 2:	size_bit=4;input_byte_f=(float*)malloc(sizeof(char)*(input_size-1024));			fread(input_byte_f,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_f;
				break;
        case 6:	size_bit=2;input_byte_u=(short*)malloc(sizeof(char)*(input_size-1024));			fread(input_byte_u,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_u;
				break;
        default:
				printf("File type error!");
    }

	fread(gain_byte,sizeof(float),(gain_f_size-1024)/4,gain_f);

	clock_t start = 0 ,finish = 0, mid = 0;
	start=clock();
	//set into place, not necesary but can save some trouble latter in dispatcher/kernel
	for(n=0; n<size_x*size_y; n++){
		gain_xy[n]=(*(float*)(gain_byte+n))*10.0;
	}
	finish=clock();
	printf("read gain_time %ds \n",(finish-start)/CLOCKS_PER_SEC);
	mid=clock();
	
	float indicator=0;
	indicator=dispatcher_gpu_syndicate( coor_xy, source, gain_xy, size_x, size_y, slice_n, file_type );

	//defect correction for points
	finish=clock();
	printf( "gain time: memcpy %ds, calc %ds\n",(indicator)/CLOCKS_PER_SEC, (finish-mid)/CLOCKS_PER_SEC );

	fclose(input);
	fclose(gain_f);
	free(gain_byte);
	switch(file_type){
        case 0:free(input_byte_c);break;
        case 1:free(input_byte_s);break;
        case 2:free(input_byte_f);break;
        case 6:free(input_byte_u);break;
        default:printf("File type error!\n");
    }

	//write output file
	output=fopen(fout,"wb");
	//write head for output file
	fwrite(head,(sizeof(MrcHeader)+head->next),1,output);
	start=clock();
		
	//fwrite(coor_xy,sizeof(float),size_x*size_y*slice_n,output);
	fwrite(coor_xy, sizeof(float)*size_x*size_y*slice_n, 1, output);
	//POTENTIAL(?): change to write all content at one time e.g.( coor_xy, sizeof(float)*size_x*size_y*slice_n, output )
		
	printf("entrice written: %ld\n", size_x*size_y*slice_n);
	printf("head length: %d\n", (sizeof(MrcHeader)+head->next));
	finish=clock();
	printf("write_time %d \n", (finish-start)/CLOCKS_PER_SEC);

	start=clock();
	free(coor_xy);
	fclose(output);
	finish=clock();
	printf("free time %ds \n",(finish-start)/CLOCKS_PER_SEC);
	printf("Defect and gain correction finished!\n");

	printf("En Taro Tassadar!!!\n");
	return 0;
}


////
/**********************/
float dispatcher_gpu_syndicate( float *coord_l, void *src,float *gain_l , long size_x, long size_y, long slice_n, int type /*, long src_size */){
	////NOTICE: src_size = sizeof( 'src_entry' )*size_x*size_y*slice_n
	//type check
	if ( type != 0 && type != 1 && type != 2 && type != 6 ) {
		printf( "dispatcher_gpu_syndicate: wrong input Mrc type: %d \n", type );
		return -1;
	}

	int step=0;
	long offset_temp=0;
	switch( type ){
		case 0:
			step = 1;
			break;
		case 1:
			step = 2;
			break;
		case 6:
			step = 2;
			break;
		case 2:
			step = 4;
			break;
		default:
			printf("What have you done?!...");
			exit(123);
	}
	long src_size = step * size_x * size_y * slice_n; 

	//set up cuda 
	hipSetDevice( GPU_DEVICE_0 );
	void *device_coord_1=NULL, *device_coord_2=NULL;
	void *device_gain_1=NULL, *device_gain_2=NULL;
	void *device_src_1 = NULL, *device_src_2 = NULL;
	int left_over = 0;
	hipError_t f1, f2, f3, f4, f5, f6;
	long long total_s=0, single_s=0, unit_n=0;
	long long offset = 0; 
	clock_t t1=0, t2=0;
	float ret = 0;

	single_s = size_x * size_y;
	total_s = single_s * slice_n;
	if ( total_s%2 != 0 ){
		left_over = 1;
	}
	offset = total_s/2;
	
	printf("Initialize allocation.\n");
	hipSetDevice( GPU_DEVICE_0 );
	f1 = hipMalloc( (void **)&device_coord_1,  sizeof(float)*total_s/2  );
	f2 = hipMalloc( (void **)&device_src_1, src_size );
	f3 = hipMemcpy( device_src_1, src, src_size, hipMemcpyHostToDevice );
	hipSetDevice( GPU_DEVICE_1 );
	f4 = hipMalloc( (void **)&device_coord_2,  sizeof(float)*(total_s/2 + left_over) );
	f5 = hipMalloc( (void **)&device_src_2, src_size );
	f6 = hipMemcpy( device_src_2, src, src_size, hipMemcpyHostToDevice );
	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess || f3 != hipSuccess || f4 != hipSuccess || f5 != hipSuccess || f6 != hipSuccess ){
		printf("cuda memory allocation failed: %s -- %s -- %s \n", f1, f2, f3 );
		exit(4);
	}
	printf("Allocation done.\n");
	
	printf("Before any Kernel, statues: leftover - %d, total_s - %d, offset - %d, src_size - %d, size_x - %d, size_y - %d, slice_n %d \n", left_over, total_s, offset, src_size, size_x, size_y, slice_n );
	t1 = clock();
	if (type != 2){
			printf(" type: %d \n", type);
			long unit_temp = ceil(total_s/(2.0*( GTHREAD_N )));
			
			offset_temp = (total_s/2)*step;
			printf("offest_temp: %d, unit_temp: %d, step: %d \n", offset_temp, unit_temp, step );

			hipSetDevice( GPU_DEVICE_0 );
			transfer_kernel<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord_1, (char *)device_src_1, total_s/2, unit_temp, step);
			hipSetDevice( GPU_DEVICE_1 );
			transfer_kernel<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord_2,((char*)device_src_2 + offset_temp), total_s/2 + left_over, unit_temp, step);

			f1 = hipDeviceSynchronize();
			cudaErrorCheck(f1, 321);
	}else{
		printf("type: 2\n");
		hipSetDevice( GPU_DEVICE_0 );
		f1 = hipMemcpy( device_coord_1, src, sizeof(float)*total_s/2, hipMemcpyHostToDevice );
		hipSetDevice( GPU_DEVICE_1 );
		f3 = hipMemcpy( device_coord_2, (((float*)src) + total_s/2), sizeof(float)*(total_s/2 + left_over), hipMemcpyHostToDevice );
		if ( f1 != hipSuccess || f3 != hipSuccess ){
			printf("purge finish-off failed: %s \n", f1 );
			exit(121);
		}
	}
	t2 = clock();
	ret = (t2-t1);
	hipSetDevice( GPU_DEVICE_0 );
	f1 = hipFree(device_src_1);
	hipSetDevice( GPU_DEVICE_1 );
	f2 = hipFree(device_src_2);
	if ( f2 != hipSuccess || f2 != hipSuccess ){
		printf("finish-off failed: %s \n", f1 );
		exit(121);
	}

	//data transfer to device
	printf("Initialize Memcpy: host->device\n");
	hipSetDevice( GPU_DEVICE_0 );
	f4 = hipMalloc( (void **)&device_gain_1,  sizeof(float)*single_s );
	f2 = hipMemcpy( device_gain_1, gain_l, sizeof(float)*single_s, hipMemcpyHostToDevice );
	//GRAM allocation check
	if (  f2 != hipSuccess || f4 != hipSuccess ){
		printf("hipMemcpy failed(H->D): %s -- %s，device: %d \n", f2, f4, GPU_DEVICE_0 );
		exit(5);
	}
	hipSetDevice( GPU_DEVICE_1 );
	f4 = hipMalloc( (void **)&device_gain_2,  sizeof(float)*single_s );
	f2 = hipMemcpy( device_gain_2, gain_l, sizeof(float)*single_s, hipMemcpyHostToDevice );
	//GRAM allocation check
	if (  f2 != hipSuccess || f4 != hipSuccess ){
		printf("hipMemcpy failed(H->D): %s -- %s, device: %d \n", f2, f4, GPU_DEVICE_1 );
		exit(5);
	}
	printf("Memcpy Done: host->device\n");
	
	//activate kernel
	unit_n = ceil(total_s/(2.0*( GTHREAD_N ))); //take notice that this "2.0" is very important to cast the calc into float
	printf("Insider: the unit_n is %d \n", unit_n);	

			printf("waypoint Serpent\n");
			hipSetDevice( GPU_DEVICE_0 );
			mutiplier_kernel_syndicate<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord_1, (float*)device_gain_1, total_s/2, single_s, unit_n, 0 ); 
			//HAZARD: slice_c might be a problem... + a offset, yes offset will do.BUT NOT FULLY VERIFIED YET
			hipSetDevice( GPU_DEVICE_1 );
			mutiplier_kernel_syndicate<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord_2, (float*)device_gain_2, total_s/2+left_over, single_s, unit_n, offset );
			printf("Serpent out\n");

	//thread synchronization 
	/*
	f1 = hipDeviceSynchronize();
	if ( f1 != hipSuccess ){
		printf("cuda sync failed: %s \n", f1 );
		exit(9);
	}
	printf("Sync Done\n");
	*/
	//data transfer from device
	///* Kernel validation
	printf("Initialize Memcpy: device->host\n");
	hipSetDevice( GPU_DEVICE_0 );
	f1 = hipMemcpy( coord_l, device_coord_1, sizeof(float)*total_s/2, hipMemcpyDeviceToHost );
	printf("Memcpy: device->host, half-way through\n");
	////Narrator: replace hipMemcpy manually,if hipMemcpy doesn't work for too long addresses
	hipSetDevice( GPU_DEVICE_1 );
	f2 = hipMemcpy( (coord_l + total_s/2), device_coord_2, sizeof(float)*(total_s/2 + left_over), hipMemcpyDeviceToHost );
	//Narrator: if manual copy doesn't work either, then we declare another agent and do a manual cascade.
	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("hipMemcpy failed(D->H): %s -- %s \n", f1, f2 );
		exit(6);
	}
	printf("Memcpy Done: device->host\n");
	
	//clean up
	printf("Initialize hipFree\n");
	hipSetDevice( GPU_DEVICE_0 );
	f1 = hipFree( device_coord_1 );
	f2 = hipFree( device_gain_1 );
	hipSetDevice( GPU_DEVICE_1 );
	f3 = hipFree( device_coord_2 );
	f4 = hipFree( device_gain_2 );
	if ( f1 != hipSuccess || f2 != hipSuccess || f3 != hipSuccess || f4 != hipSuccess ){
		printf( "hipFree failed: %s -- %s -- %s -- %s\n", f1, f2, f3, f4 );
		exit(7);
	}

	printf("hipFree Done\n");	
	printf("Dispatcher out\n");
	return ret;
}

__global__ void mutiplier_kernel_syndicate(float *coord, float *gain, long long total_s, long long single_s, long long unit_n, long long offset){
	//execute all those mutiplications of type_c data
	long long index=0;
	long long i = 0; 
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long long slice_c=0;

	for (i=0; i<unit_n; i++){
		if (index*unit_n + i >= total_s ){
			return;
		}
		slice_c = (index*unit_n+i+offset)%(single_s);
		coord[index*unit_n+i] = (coord[index*unit_n+i] * gain[slice_c]);
	}	
	return;
}

__global__ void transfer_kernel( float *coord, char *src, long long total_s, long long unit_n, int step ){
	long long index = 0;
	long long i=0;
	index = blockDim.x*blockIdx.x + threadIdx.x;

	for ( i=0; i<unit_n; i++ ){
		if ( index*unit_n+i >= total_s ){
			return;
		}
		coord[index*unit_n+i] = *(src + (index*unit_n+i)*step);
	}
	return;
}

bool cudaErrorCheck( hipError_t x, int error_code ){
	if ( x != hipSuccess ){
		exit(error_code);
	}
	return true;
}

bool alpha_test(){
	char *log_name = "tsp_alpha_test.log";
	FILE *log_f = NULL, *ori_f = NULL, *out_f = NULL;
	
	float source[SIZE] = {0};
	float gain[SIZE/2] = {0};
	float des[SIZE] = {0};
	float sum_check = 0;
	int x_size = 10;
	long i=0, j=0;
	bool flag = true;
	int misfit_fh = 0, misfit_lh = 0;

	//initialization
	for ( j=0; j<SIZE; j++ ) {
		if ( j < SIZE/2 ){
			source[j] = 2;
			gain[j] = 2;
		}
		else if ( j < SIZE ){
			source[j] = 4;
		}
	}

	/*
	float dispatcher_gpu_syndicate(float *coord_l, void *src, float *gain_l , long size_x, long size_y, long slice_n, int type, long );
	*/
	dispatcher_gpu_syndicate( des, source, gain, SIZE/(2*x_size), x_size, 2, 2/*,SIZE*4*/ );
	
	//validation
	for ( i=0; i<SIZE; i++ ) {
		if ( i <SIZE/2 && des[i] != 4){
			flag = false;
			misfit_fh += 1;
		}
		else if( i >= SIZE/2 && i < SIZE && des[i] != 8 ){
			flag = false;
			misfit_lh += 1;
		}
		sum_check += des[i];
		continue;
	}
	
	printf( "sum_check: %f \n", sum_check );
	printf("Misfit, first-half: %d; last-half: %d \n", misfit_fh, misfit_lh );
	log_f = fopen( log_name, "w");
	ori_f = fopen( "tsp_alpha_ori.log", "w" );
	out_f = fopen( "tsp_alpha_out.log", "w" );

	for ( j=0; j<SIZE ; j++){
		fprintf( log_f, "%.0f ", des[j]-2*source[j] );
		fprintf( ori_f, "%.0f ", source[j] );
		fprintf( out_f, "%.0f ", des[j] );
	}
	fclose( log_f );
	fclose( ori_f );
	fclose( out_f );
	printf( "--" );
	return flag;
}
bool beta_test( int type ){
	if ( type != 0 && type != 1 && type != 6 ){
		printf("Wrong type: %d \n", type);
		return false;
	}

	char *log_name = "tsp_beta_test.log";
	FILE *log_f = NULL, *ori_f = NULL, *out_f = NULL;
	char source[B_SIZE] = {0};
	float gain[B_GAIN_S] = {0};
	float des[B_SIZE] = {0};
	long i=0, j=0;
	bool flag = true;
	int misfit_fh = 0, misfit_lh = 0;

	//initialization
	for (j = 0; j < B_GAIN_S; j++ ){
		gain[j] = 2;
	}

	for ( j=0; j<B_SIZE; j++ ) {
		if ( j < B_SIZE/2 ){
			source[j] = '!';
		}
		else{
			source[j] = '#';
		}
		des[j] = -1;
	}

	/*
	float dispatcher_gpu_syndicate(float *coord_l, void *src, float *gain_l , long size_x, long size_y, long slice_n, int type, long );
	*/
	dispatcher_gpu_syndicate( des, source, gain, 75, 75, 5, type/*, 75*75*5*sizeof(char)*/ );
	
	//validation
	for ( i=0; i<B_SIZE; i++ ) {
		if (  i< B_SIZE/2 && (int)des[i] != 66){
			flag = false;
			misfit_fh += 1;
		}
		else if ( i >= B_SIZE/2 && (int)des[i] != 70 ){
			flag = false;
			misfit_lh += 1;
		}
	}
	
	printf("Misfit, first-half: %d, last-half: %d \n", misfit_fh, misfit_lh );
	log_f = fopen( log_name, "w");
	ori_f = fopen( "tsp_beta_ori.log", "w" );
	out_f = fopen( "tsp_beta_out.log", "w" );

	for ( j=0; j<B_SIZE ; j++){
		fprintf( log_f, "%.0f ", des[j]-2*(int)source[j] );
		fprintf( ori_f, "%c ", source[j] );
		fprintf( out_f, "%d ", (int)des[j] );
	}
	fclose( log_f );
	fclose( ori_f );
	fclose( out_f );
	printf( "--" );
	return flag;
}

////
/**********************/