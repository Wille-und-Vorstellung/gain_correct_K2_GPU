#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "ReadFile.h"
#include "ReadFile.c"
#define LINE 1024
#define NAME 1024
#include "time.h"
#include "omp.h"
#include "hip/hip_runtime.h"

/*****Just DON'T touch any of those Macros below ok, unless you UNDERSTAND what you're doing, which means refering to the context where them're used *****/
#define GRID_BLOCK 16 //16
#define BLOCK_SIZE 32 //32
#define GTHREAD_N ( GRID_BLOCK * BLOCK_SIZE )
#define GIGA 1073741824
#define MP_THREAD 10
#define TEST_RUN false

#define B_SIZE 28125 // debug only, beta_test()
#define B_GAIN_S 5625

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads);

__global__ void mutiplier_kernel_8bit(char *coord, float *gain, long long, long long, long long, long long);

__global__ void transfer_kernel( char *coord, char *src, long long total_s, long long unit_n, int step );

float dispatcher_gpu_8bit(char *coord_l, void *src, float *gain_l , long size_x, long size_y, long slice_n, int type, long );

bool cudaErrorCheck( hipError_t, int );

bool beta_test( int type = 0 ); //type 0

int main(int argc, char *argv[]){
	clock_t start_ts = 0, end_ts = 0;

	start_ts = clock();
	bool h = true;
	if ( TEST_RUN ){
		h = beta_test();
		if ( h == false ){
			printf( "Beta Failed \n" );
			return 2;
		}
		printf( "Passed \n" );
		return 0;
	}

    char *raw_name,*out_name,*gain_name;
    FILE *file,*gain;
    //int threads;
    MrcHeader *inhead;

    raw_name=(char *)malloc(NAME*sizeof(char));
    out_name=(char *)malloc(NAME*sizeof(char));
    gain_name=(char *)malloc(NAME*sizeof(char));

    inhead=(MrcHeader *)malloc(sizeof(MrcHeader));

    if(argc!=4)
    {
        printf("Please input: raw_image gain_name out_name threads\n");
        return;
    }
    raw_name=argv[1];
    gain_name=argv[2];
    out_name=argv[3];
    //threads=atoi(argv[4]);

    file=fopen(raw_name,"rb");	
    mrc_read_head(file,inhead);
    fclose(file);

	//printf("raw_name %s, out_name %s, gain_name %s\n",raw_name,out_name,gain_name);
    defect_gain_correct(raw_name,gain_name,out_name,inhead, MP_THREAD );
	/*
	free(raw_name);
	free(out_name);
	free(gain_name);
	*/
	end_ts = clock();
	printf( "Total times: %ds \n", (end_ts - start_ts)/CLOCKS_PER_SEC );
	
	return 0;
}

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads){
	unsigned char lbuf[LINE];
	float fbuf[LINE];
	float *tmp_array,*gain_xy=NULL;
	//unsigned char *coor_xy;
	char *coor_xy=NULL;
	int i,j,x,y,p,k,w,w_row,w_length,m,range,point_s,point_e,n_file,size_bit,*pxy_array;
	long int n;
	FILE *input,*output,*gain_f, *temp_head;
	char *input_byte_c=NULL;short *input_byte_s=NULL;float *input_byte_f=NULL;short *input_byte_u=NULL;
	srand((unsigned) time(NULL));
	void *source =NULL;

	//open input output and gain file
	printf("Start defect and gain correction|\ninput %s output %s gain %s\n",fin,fout,gain);
	input=fopen(fin,"rb");
	gain_f=fopen(gain,"rb");

	//revise input image header then written for output file
	int size_x=head->nx;
	int size_y=head->ny;
	int slice_n=head->nz;
    int file_type=head->mode;
	head->mode=0;
        switch(file_type)
                {
                case 0:size_bit=1;break;
                case 1:size_bit=2;break;
                case 2:size_bit=4;break;
                case 6:size_bit=2;break;
                default:printf("File type error!");
                }

	//calculate the size of the input and gain file by byte.
	fseek(input,0,SEEK_END);
	long int input_size=ftell(input);
	rewind(input);

	fseek(gain_f,0,SEEK_END);
	long int gain_f_size=ftell(gain_f);
	rewind(gain_f);

	//skip header
	fseek(input,1024,0);
	fseek(gain_f,1024,0);
	//printf("input_size %d file_type %d size_bit %d\n",input_size,file_type,size_bit);

	//malloc memory for the file pointer
	float *gain_byte=(float*)malloc(sizeof(char*)*(gain_f_size-1024));

	//malloc memory for pxy_array
        pxy_array=(int*)malloc(sizeof(int*)*4000);

	//malloc memory for variate 
	tmp_array=(float*)malloc(sizeof(float)*size_x);

	printf("pointer check(pre-allocate): %p -- %p \n", coor_xy, gain_xy);
	gain_xy=(float*)malloc(sizeof(float)*size_y*size_x);
	//coor_xy=(unsigned char*)malloc(sizeof(unsigned char)*size_y*size_x*slice_n);
	coor_xy=(char*)malloc(sizeof(char)*size_y*size_x*slice_n);
	printf("pointer check(post-allocate): %p -- %p \n", coor_xy, gain_xy);

	//read the input and gain file into memory
	//note: the byte size(second parameter) is basen on the type of the file
	printf( "--> Input Mrc type: %d \n", file_type );
	printf("Before dispatch, statues: src_size - %d, size_x - %d, size_y - %d, slice_n %d \n", input_size-1024, size_x, size_y, slice_n );

	switch(file_type)
                {
                case 0:size_bit=1;input_byte_c=(char*)malloc(sizeof(char)*(input_size-1024));fread(input_byte_c,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_c;
				break;
                case 1:size_bit=2;input_byte_s=(short*)malloc(sizeof(char)*(input_size-1024));fread(input_byte_s,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_s;
				break;
                case 2:size_bit=4;input_byte_f=(float*)malloc(sizeof(char)*(input_size-1024));fread(input_byte_f,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_f;
				break;
                case 6:size_bit=2;input_byte_u=(short*)malloc(sizeof(char)*(input_size-1024));fread(input_byte_u,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_u;
				break;
                default:printf("File type error!");
                }

	fread(gain_byte,sizeof(float),(gain_f_size-1024)/4,gain_f);

	//read gain 
	float badcut=6.0;
	long int num_gain=0;
	int defect_num=0;
	clock_t start = 0 ,finish = 0, mid = 0;

	long int total_num=0,gain_num=0,gain_m=0;
	
	start=clock();
	for(n=0; n<size_x*size_y; n++)
		{
		gain_xy[n]=(*(float*)(gain_byte+n))*10.0;
		}
	finish=clock();
	printf("read gain_time %ds \n",(finish-start)/CLOCKS_PER_SEC);
	mid=clock();
	
	float indicator=0;
	//printf("pointer check(pre-invoke): %p -- %p \n", coor_xy, gain_xy);
	indicator=dispatcher_gpu_8bit( coor_xy, source, gain_xy, size_x, size_y, slice_n, file_type, input_size-1024 );

	//defect correction for points
	finish=clock();
	printf( "gain time: memcpy %ds, calc %ds\n",(indicator)/CLOCKS_PER_SEC, (finish-mid)/CLOCKS_PER_SEC );

	fclose(input);
	fclose(gain_f);
	free(gain_byte);
	switch(file_type)
               {
               case 0:free(input_byte_c);break;
               case 1:free(input_byte_s);break;
               case 2:free(input_byte_f);break;
               case 6:free(input_byte_u);break;
               default:printf("File type error!\n");
               }
	free(tmp_array);


	//write output file
	output=fopen(fout,"wb");
	//write head for output file
	fwrite(head,(sizeof(MrcHeader)+head->next),1,output);
	start=clock();
		{
		
		//fwrite(coor_xy,sizeof(char),size_x*size_y*slice_n,output);
		fwrite(coor_xy, sizeof(char)*size_x*size_y*slice_n, 1, output);
		}
		printf("entrice written: %ld\n", size_x*size_y*slice_n);
		printf("head length: %d\n", (sizeof(MrcHeader)+head->next));
	finish=clock();
	printf("write_time %d \n",finish-start);

	start=clock();
	free(coor_xy);
	fclose(output);
	finish=clock();
	printf("free time %ds \n",(finish-start)/CLOCKS_PER_SEC);
	printf("Defect and gain correction finished!\n");

	printf("En Taro Tassadar!!!\n");
	return 0;
}


////
/**********************/
float dispatcher_gpu_8bit( char *coord_l, void *src,float *gain_l , long size_x, long size_y, long slice_n, int type, long src_size ){
	////NOTICE: src_size = sizeof( 'src_entry' )*size_x*size_y*slice_n
	//type check	
	if ( type != 0 && type != 1 && type != 2 && type != 6 ) {
		printf( "dispatcher_gpu_8bit: wrong input Mrc type: %d \n", type );
		return -1;
	}

	//set up cuda 
	hipSetDevice(0);	
	void *device_coord_1=NULL, *device_coord_2=NULL;
	void *device_gain_1=NULL, *device_gain_2=NULL;
	void *device_src=NULL;
	int left_over = 0;
	hipError_t f1, f2, f3, f4;
	long long total_s=0, single_s=0, unit_n=0;
	long long offset = 0; 
	clock_t t1=0, t2=0;
	float ret = 0;

	single_s = size_x * size_y;
	total_s = single_s * slice_n;
	if ( total_s%2 != 0 ){
		left_over = 1;
	}
	offset = total_s/2;
	
	printf("Initialize allocation.\n");
	printf("Required total(GB): %ld \n", (sizeof(float)*total_s + sizeof(float)*single_s)/GIGA);
	printf("Required each: %ld -- %ld \n", sizeof(float)*total_s, sizeof(float)*single_s);
	f1 = hipMalloc( (void **)&device_coord_1,  sizeof(char)*total_s/2  );
	f2 = hipMalloc( (void **)&device_src, src_size );
	f4 = hipMemcpy( device_src, src, src_size, hipMemcpyHostToDevice );

	f3 = hipMalloc( (void **)&device_coord_2,  sizeof(char)*(total_s/2 + left_over)  );
	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess || f3 != hipSuccess || f4 != hipSuccess ){
		printf("cuda memory allocation failed: %s -- %s -- %s \n", f1, f2, f3 );
		exit(4);
	}
	printf("Allocation done.\n");
	
	printf("Before any Kernel, statues: leftover - %d, total_s - %d, offset - %d, src_size - %d, size_x - %d, size_y - %d, slice_n %d \n", left_over, total_s, offset, src_size, size_x, size_y, slice_n );
	t1 = clock();
	if (type != 0){ 
			printf(" type: %d \n", type);
			long unit_temp = ceil(total_s/(2.0*( GTHREAD_N )));
			int step=0;
			long offset_temp=0;
			switch( type ){
				case 2:
					step = 4;
					break;
				case 1:
					step = 2;
					break;
				case 6:
					step = 2;
					break;
				default:
					printf("What have you done?!...");
					exit(123);
			}

			offset_temp = (total_s/2)*step;
			printf("offest_temp: %d, unit_temp: %d, step: %d \n", offset_temp, unit_temp, step );
			transfer_kernel<<< GRID_BLOCK, BLOCK_SIZE >>>( (char*)device_coord_1, (char*)device_src, total_s/2, unit_temp, step);

			transfer_kernel<<< GRID_BLOCK, BLOCK_SIZE >>>( (char*)device_coord_2,((char*)device_src + offset_temp), total_s/2 + left_over, unit_temp, step);

			f1 = hipDeviceSynchronize();
			cudaErrorCheck(f1, 321);
	}else{
		printf("type: 0\n");
		f1 = hipMemcpy( device_coord_1, src, sizeof(char)*total_s/2, hipMemcpyHostToDevice );
		f3 = hipMemcpy( device_coord_2, (((char*)src) + total_s/2), sizeof(char)*(total_s/2 + left_over), hipMemcpyHostToDevice );
		if ( f1 != hipSuccess || f3 != hipSuccess ){
			printf("purge finish-off failed: %s \n", f1 );
			exit(121);
		}
	}
	t2 = clock();
	ret = (t2-t1);
	f2 = hipFree(device_src);
	if ( f2 != hipSuccess ){
		printf("purge finish-off failed: %s \n", f1 );
		exit(121);
	}

	//end{mem transfer}

	//data transfer to device
	printf("Initialize Memcpy: host->device\n");
	f4 = hipMalloc( (void **)&device_gain_1,  sizeof(float)*single_s );
	//f1 = hipMemcpy( device_coord_1, coord_l, sizeof(float)*total_s/2, hipMemcpyHostToDevice );
	f2 = hipMemcpy( device_gain_1, gain_l, sizeof(float)*single_s, hipMemcpyHostToDevice );
	//f3 = hipMemcpy( device_coord_2, (coord_l + total_s/2), sizeof(float)*(total_s/2 + left_over), hipMemcpyHostToDevice );
	//GRAM allocation check
	if (  f2 != hipSuccess || f4 != hipSuccess ){
		printf("hipMemcpy failed(H->D): %s -- %s \n", f2, f4);
		exit(5);
	}
	//*/
	printf("Memcpy Done: host->device\n");
	
	//activate kernel
	unit_n = ceil(total_s/(2.0*( GTHREAD_N ))); 
	printf("Insider: the unit_n is %d \n", unit_n);	

			printf("waypoint Serpent\n");

			mutiplier_kernel_8bit<<< GRID_BLOCK, BLOCK_SIZE >>>( (char*)device_coord_1, (float*)device_gain_1, total_s/2, single_s, unit_n, 0 ); 
			
			f1 = hipDeviceSynchronize();
			if ( f1 != hipSuccess ){
				printf("cuda sync(mid-way) failed: %s \n", f1 );
				exit(12);
			}
			
			mutiplier_kernel_8bit<<< GRID_BLOCK, BLOCK_SIZE >>>( (char*)device_coord_2, (float*)device_gain_1, total_s/2+left_over, single_s, unit_n, offset );

			printf("Serpent out\n");

	//thread synchronization 
	f1 = hipDeviceSynchronize();
	if ( f1 != hipSuccess ){
		printf("cuda sync failed: %s \n", f1 );
		exit(9);
	}
	printf("Sync Done\n");
	
	//data transfer from device
	///* Kernel validation
	printf("Initialize Memcpy: device->host\n");
	
	f1 = hipMemcpy( coord_l, device_coord_1, sizeof(char)*total_s/2, hipMemcpyDeviceToHost );
	printf("Memcpy: device->host, half-way through\n");
	////Narrator: replace hipMemcpy manually,if hipMemcpy doesn't work for too long addresses
	f2 = hipMemcpy( (coord_l + total_s/2), device_coord_2, sizeof(char)*(total_s/2 + left_over), hipMemcpyDeviceToHost );
	//Narrator: if manual copy doesn't work either, then we declare another agent and do a manual cascade.

	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("hipMemcpy failed(D->H): %s -- %s \n", f1, f2 );
		exit(6);
	}
	printf("Memcpy Done: device->host\n");
	
	//clean up
	printf("Initialize hipFree\n");
	f1 = hipFree( device_coord_1 );
	f2 = hipFree( device_gain_1 );
	f3 = hipFree( device_coord_2 );
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("hipFree failed: %s -- %s \n", f1, f2);
		exit(7);
	}
	printf("hipFree Done\n");
	
	printf("Dispatcher out\n");
	return ret;
}

__global__ void mutiplier_kernel_8bit(char *coord, float *gain, long long total_s, long long single_s, long long unit_n, long long offset){
	//execute all those mutiplications of type_c data
	long long index=0;
	long long i = 0; 
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long long slice_c=0;

	for (i=0; i<unit_n; i++){
		if (index*unit_n + i >= total_s ){
			return;
		}
		slice_c = (index*unit_n+i+offset)%(single_s);
		//printf("In M kernel: %d, gain %d at %d \n", coord[index*unit_n+i], gain[slice_c], index*unit_n+i );
		coord[index*unit_n+i] = (coord[index*unit_n+i] * gain[slice_c]);
		//printf("In M kernel: %d \n", coord[index*unit_n+i] );
	}
	
	return;
}

__global__ void transfer_kernel( char *coord, char *src, long long total_s, long long unit_n, int step ){
	long long index = 0;
	long long i=0;
	index = blockDim.x*blockIdx.x + threadIdx.x;

	for ( i=0; i<unit_n; i++ ){
		if ( index*unit_n+i >= total_s ){
			return;
		}

		coord[index*unit_n+i] = *(src + (index*unit_n+i)*step);
		//printf("In transfer kernel: %d at \n", coord[index*unit_n+i], index*unit_n+i );

	}

	return;
}

bool cudaErrorCheck( hipError_t x, int error_code ){
	if ( x != hipSuccess ){
		exit(error_code);
	}
	return true;
}

bool beta_test( int type ){
	if ( type != 0 && type != 1 && type != 6 ){
		printf("Wrong type: %d \n", type);
		return false;
	}

	char *log_name_test = "t8_beta_test.log";
	char *log_name_ori = "t8_beta_ori.log";
	char *log_name_out = "t8_beta_out.log";

	FILE *log_f = NULL, *ori_f = NULL, *out_f = NULL;
	
	char source[B_SIZE] = {0};
	float gain[B_GAIN_S] = {0};
	char des[B_SIZE] = {0};
	float sum_check = 0;

	int x_size = 10;
	int slice_N = 2;

	long i=0, j=0;
	bool flag = true;
	int misfit_fh = 0, misfit_lh = 0;

	//initialization
	for (j = 0; j < B_GAIN_S; j++ ){
		gain[j] = 2;
	}

	for ( j=0; j<B_SIZE; j++ ) {
		if ( j < B_SIZE/2 ){
			source[j] = '!';
		}
		else{
			source[j] = '#';
		}
		des[j] = -1;
	}

	/*
	float dispatcher_gpu_8bit(float *coord_l, void *src, float *gain_l , long size_x, long size_y, long slice_n, int type, long );
	*/
	//dispatcher_gpu_8bit( des, source, gain, B_SIZE/(slice_N*x_size), x_size, slice_N, type, B_SIZE*sizeof(char) );
	dispatcher_gpu_8bit( des, source, gain, 75, 75, 5, type, 75*75*5*sizeof(char) );
	
	//validation
	for ( i=0; i<B_SIZE; i++ ) {
		if (  i< B_SIZE/2 && (int)des[i] != 66){
			flag = false;
			misfit_fh += 1;
		}
		else if ( i >= B_SIZE/2 && (int)des[i] != 70 ){
			flag = false;
			misfit_lh += 1;
		}
	}
	
	printf("Misfit, first-half: %d, last-half: %d \n", misfit_fh, misfit_lh );
	log_f = fopen( log_name_test, "w");
	ori_f = fopen( log_name_ori, "w" );
	out_f = fopen( log_name_out, "w" );

	for ( j=0; j<B_SIZE ; j++){
		fprintf( log_f, "%d ", des[j]-2*(int)source[j] );
		fprintf( ori_f, "%c ", source[j] );
		fprintf( out_f, "%d ", (int)des[j] );
	}
	fclose( log_f );
	fclose( ori_f );
	fclose( out_f );
	printf( "--\n" );
	return flag;
}

////
/**********************/