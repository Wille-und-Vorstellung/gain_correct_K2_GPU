#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "ReadFile.h"
#include "ReadFile.c"
#define LINE 1024
#define NAME 1024
#include "time.h"
#include "omp.h"
#include "hip/hip_runtime.h"

#define GRID_BLOCK 1
#define BLOCK_SIZE 1
#define GTHREAD_N ( GRID_BLOCK * BLOCK_SIZE )
#define GIGA 1073741824
#define MP_THREAD 10

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads);
int dispatcher_gpu(float *coord_l, float *gain_l , long size_x, long size_y, long slice_n, int type);

__global__ void mutiplier_kernel_type_c_exp_split(float *coord, float *gain, long long, long long, long long, long long);
void mutiplier_kernel_type_c_exp_test(float *coord, float *gain, long long total_s, long long single_s, long long unit_n);

__global__ void mutiplier_kernel_type_c(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n);
__global__ void mutiplier_kernel_type_su(float *coord, float *gain, void* src,long size_x, long size_y, long slice_n);
__global__ void mutiplier_kernel_type_f(float *coord, float *gain, void* src,long size_x, long size_y, long slice_n);
void mutiplier_kernel_test(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n);

int main(int argc, char *argv[])
{

	//printf("raw_name %s, out_name %s, gain_name %s\n",raw_name,out_name,gain_name);
	char *raw_name = NULL, *gain_name = NULL, *out_name = NULL;
	MrcHeader *inhead=NULL;
    defect_gain_correct(raw_name,gain_name,out_name,inhead, MP_THREAD );
	/*
	free(raw_name);
	free(out_name);
	free(gain_name);
	*/
	return 0;
}

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads){
	
	//revise input image header then written for output file
	int size_x=5000;
	int size_y=200;
	int slice_n=2;
    int indicator=0;
	int result=0;
	int sum=0;
	float *coor_xy = NULL;
	float *gain_xy = NULL;

	coor_xy = (float *)malloc( sizeof(int)*size_x*size_y*slice_n );
	gain_xy = (float *)malloc( sizeof(int)*size_x*size_y );

	for (int j=0; j < size_x*size_y; j++ ){
		gain_xy[j] = 2;
	}
	for (int k=0; k < size_x*size_y*slice_n; k++ ){
		if ( k < size_x*size_y  ) {
			coor_xy[k] = 1;
		}
		else {
			coor_xy[k] = 2;
		}
	}

	indicator=dispatcher_gpu( coor_xy, gain_xy, size_x, size_y, slice_n, 0 );

	for(int i=0; i<size_x*size_y*slice_n; i++){
		if ( i < size_x*size_y && coor_xy[i] != 2 ) {
			result+=1;
		}
		else if( i > size_x*size_y && coor_xy[i] != 4 ) {
			result+=1;
		}
		else;
		sum += coor_xy[i];
	}

	printf("result: %ld, %d\n", result, sum );
	return 0;
}

/**********************/

int dispatcher_gpu(float *coord_l, float *gain_l , long size_x, long size_y, long slice_n, int type){
	//set up cuda 
	hipSetDevice(0);	
	void *device_coord_1=NULL, *device_coord_2=NULL;
	void *device_gain_1=NULL, *device_gain_2=NULL;
	int left_over = 0;
	hipError_t f1, f2, f3, f4;
	long long total_s=0, single_s=0, unit_n=0;
	long long offset = 0; 

	single_s = size_x * size_y;
	total_s = single_s * slice_n;
	if ( total_s%2 != 0 ){
		left_over = 1;
	}
	offset = total_s/2;
	/*
	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);
	*/
	printf("Initialize allocation.\n");
	printf("Required total(GB): %ld \n", (sizeof(float)*total_s + sizeof(float)*single_s)/GIGA);
	printf("Required each: %ld -- %ld \n", sizeof(float)*total_s, sizeof(float)*single_s);
	f1 = hipMalloc( (void **)&device_coord_1,  sizeof(float)*total_s/2  );
	f2 = hipMalloc( (void **)&device_gain_1,  sizeof(float)*single_s );
	f3 = hipMalloc( (void **)&device_coord_2,  sizeof(float)*(total_s/2 + left_over)  );
	//f4 = hipMalloc( (void **)&device_gain_2,  sizeof(float)*single_s );
	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess || f3 != hipSuccess ){
		printf("cuda memory allocation failed: %s -- %s -- %s \n", f1, f2, f3 );
		exit(4);
	}
	printf("Allocation done.\n");
	printf("status: %s -- %s -- %s \n", f1, f2, f3);
	/*
	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);
	*/
	//data transfer to device
	printf("Initialize Memcpy: host->device\n");
	
	///* Kernel validation
	f1 = hipMemcpy( device_coord_1, coord_l, sizeof(float)*total_s/2, hipMemcpyHostToDevice );
	f2 = hipMemcpy( device_gain_1, gain_l, sizeof(float)*single_s, hipMemcpyHostToDevice );
	f3 = hipMemcpy( device_coord_2, (coord_l + total_s/2), sizeof(float)*(total_s/2 + left_over), hipMemcpyHostToDevice );
	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("hipMemcpy failed(H->D): %s -- %s \n", f1, f2);
		exit(5);
	}
	//*/
	printf("Memcpy Done: host->device\n");
	/*
	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);
	*/
	
	//activate kernel
	unit_n = (total_s/(2*( GTHREAD_N ))) + 1; 
	


			printf("waypoint C\n");

			mutiplier_kernel_type_c_exp_split<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord_1, (float*)device_gain_1, total_s/2, single_s, unit_n, 0 ); 
			//HAZARD: slice_c might be a problem... + a offset, yes offset will do.BUT NOT FULLY VERIFIED YET
			//can switch cpu here and remove the sync below
			///*
			f1 = hipDeviceSynchronize();
			if ( f1 != hipSuccess ){
				printf("cuda sync(mid-way) failed: %s \n", f1 );
				exit(12);
			}
			//*/
			mutiplier_kernel_type_c_exp_split<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord_2, (float*)device_gain_1, total_s/2+left_over, single_s, unit_n, offset );


			printf("C out\n");



	//thread synchronization 
	f1 = hipDeviceSynchronize();
	if ( f1 != hipSuccess ){
		printf("cuda sync failed: %s \n", f1 );
		exit(9);
	}
	printf("Sync Done\n");
	/*
	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);
	*/
	//data transfer from device
	///* Kernel validation
	printf("Initialize Memcpy: device->host\n");
	
	f1 = hipMemcpy( coord_l, device_coord_1, sizeof(float)*total_s/2, hipMemcpyDeviceToHost );
	printf("Memcpy: device->host, half-way through\n");
	////Narrator: replace hipMemcpy manually,if hipMemcpy doesn't work for too long addresses
	f2 = hipMemcpy( (coord_l + total_s/2), device_coord_2, sizeof(float)*(total_s/2 + left_over), hipMemcpyDeviceToHost );
	/*
	for (long long i=0; i < sizeof(float)*total_s/2 + left_over; i++){
		coord_l[i+total_s/2] = device_coord_2[i];
	}
	*/
	//Narrator: if manual copy doesn't work either, then we declare another agent and do a manual cascade.

	//GRAM allocation check
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("hipMemcpy failed(D->H): %s -- %s \n", f1, f2 );
		exit(6);
	}
	printf("Memcpy Done: device->host\n");
	/*
	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);
	*/
	
	//clean up
	printf("Initialize hipFree\n");
	f1 = hipFree( device_coord_1 );
	f2 = hipFree( device_gain_1 );
	f3 = hipFree( device_coord_2 );
	if ( f1 != hipSuccess || f2 != hipSuccess ){
		printf("hipFree failed: %s -- %s \n", f1, f2);
		exit(7);
	}
	printf("hipFree Done\n");
	/*
	printf("pointer check(host): %p -- %p \n", coord_l, gain_l);
	printf("pointer check(device): %p -- %p \n", device_coord, device_gain);
	*/
	
	printf("Dispatcher out\n");
	return 117;
}

__global__ void mutiplier_kernel_type_c_exp_split(float *coord, float *gain, long long total_s, long long single_s, long long unit_n, long long offset){
	//execute all those mutiplications of type_c data
	long long index=0;
	long long i = 0; 
	//long long unit_n = (size_x*size_y*slice_n/( GTHREAD_N ))+1;
	//long long unit_n = unit_s;
	//long unit_n = UNIT_N;
	//long unit_n = (size_x*size_y*slice_n);
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long long slice_c=0;

	printf("index check: %d\n ", index);
	//printf("Size check: %d \n", size_x*size_y*slice_n);
	printf("Size check: %d \n", total_s);
	printf("unit_n check: %d \n", unit_n);
	printf("input check: %p -- %p \n", coord, gain);

	for (i=0; i<unit_n; i++){
		//printf("head-> %d\n", i);
		if (index*unit_n + i >= total_s ){ //boundary check
			printf("X: %d\n", index*unit_n + i); //POTENTIAL: return a counter through pointer
			return;
		}
		//printf("progress: %d \n", i);
		//slice_c = (index*unit_n+i)%(size_x*size_y);
		slice_c = (index*unit_n+i+offset)%(single_s);
		coord[index*unit_n+i] = (coord[index*unit_n+i] * gain[slice_c]);
		//printf("tail-> %d\n", i);
	}
	//__threadfence()
	printf("Kernel out: %d\n", i);
	return;
}

void mutiplier_kernel_type_c_exp_test(float *coord, float *gain, long long total_s, long long single_s, long long unit_n){
	//execute all those mutiplications of type_c data
	long long index=0; 
	//long long unit_n = (size_x*size_y*slice_n/( GTHREAD_N ))+1;
	//long long unit_n = unit_s;
	//long unit_n = UNIT_N;
	//long unit_n = (size_x*size_y*slice_n);
	//index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	printf("index check: %d\n ", index);
	//printf("Size check: %d \n", size_x*size_y*slice_n);
	printf("Size check: %d \n", total_s);
	printf("unit_n check: %d \n", unit_n);

	for (int i=0; i<unit_n; i++){
		//printf("head-> %d\n", i);
		if (index*unit_n + i >= total_s ){ //boundary check
			printf("X: %d", index*unit_n + i);
			return;
		}
		//printf("progress: %d \n", i);
		//slice_c = (index*unit_n+i)%(size_x*size_y);
		slice_c = (index*unit_n+i)%(single_s);
		coord[index*unit_n+i] = (coord[index*unit_n+i] * gain[slice_c]);
		//printf("tail-> %d\n", i);
	}
	//__threadfence()
	printf("Kernel out\n");
	return;
}

__global__ void mutiplier_kernel_type_c(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n){
	//execute all those mutiplications of type_c data
	long index=0; 
	//long unit_n = (size_x*size_y*slice_n/(32*64))+1;
	//long unit_n = UNIT_N;
	long unit_n = (size_x*size_y*slice_n);
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	printf("index %d\n ", index);
	printf("Size check: %d \n", size_x*size_y*slice_n);
	printf("unit_n check: %d \n", unit_n);

	for (int i=0; i<unit_n; i++){
		printf("head %d\n", i);
		if (index*unit_n + i >= size_x*size_y*slice_n ){ //boundary check
			printf("X: %d", index*unit_n + i);
			return;
		}
		printf("progress: %d \n", i);
		//slice_c = (index*unit_n+i)%(size_x*size_y);
		coord[index*unit_n+i] = (float)(*(unsigned char*)(((unsigned char*)src)+index*unit_n+i)) * gain[slice_c];
		printf("tail %d\n", i);
	}
	//__threadfence()
	printf("Kernel out\n");
	return;
}
__global__ void mutiplier_kernel_type_su(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n){//execute all those mutiplications of type_s data
	//execute all those mutiplications of type_c data
	long index=0; 
	long unit_n = size_x*size_y*slice_n/GRID_BLOCK;
	//long unit_n = UNIT_N;
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	for (int i=0; i<unit_n; i++){
		if (index*unit_n + i >= size_x*size_y*slice_n ){ //boundary check
			return;
		}
		slice_c = (index*unit_n+i)%(size_x*size_y);
		coord[index*unit_n+i] = (*(((short*)src)+index*unit_n+i))*gain[slice_c];
	}
	//__threadfence()
	return;
}
__global__ void mutiplier_kernel_type_f(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n){//execute all those mutiplications of type_f data
	//execute all those mutiplications of type_c data
	long index=0; 
	long unit_n = size_x*size_y*slice_n/GRID_BLOCK;
	//long unit_n = UNIT_N;
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	for (int i=0; i<unit_n; i++){
		if (index*unit_n + i >= size_x*size_y*slice_n ){ //boundary check
			return;
		}
		slice_c = (index*unit_n+i)%(size_x*size_y);
		coord[index*unit_n+i] = (*(((float*)src)+index*unit_n+i))*gain[slice_c];
	}
	//__threadfence()
	return;
}

void mutiplier_kernel_test(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n){
	///////////////////////////////
	printf("Activating kernel test\n");
	//execute all those mutiplications of type_c data
	long index=0; 
	//long unit_n = size_x*size_y*slice_n/GRID_BLOCK;
	//long unit_n = UNIT_N;
	long unit_n = size_x*size_y*slice_n;
	//index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	for (int i=0; i<unit_n; i++){
		
		if (index*unit_n + i >= size_x*size_y*slice_n ){ //boundary check
			return;
		}
		
		slice_c = (index*unit_n+i)%(size_x*size_y);
		//slice_c = (i)%(size_x*size_y);
		coord[index*unit_n+i] = (*(((unsigned char*)src)+index*unit_n+i))*gain[slice_c];
		//coord[i] = (*(((unsigned char*)src)+i))*gain[slice_c];
	}
	//__threadfence()
	printf("Kernel test done\n");
	return;
}
