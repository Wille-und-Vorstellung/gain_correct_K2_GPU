#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "ReadFile.h"
#include "ReadFile.c"
#define LINE 1024
#define NAME 1024
#include "time.h"
//#include "omp.h"
#include "hip/hip_runtime.h"

#define GRID_BLOCK 32 
#define BLOCK_SIZE 64
#define UNIT_N 1024*1024

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads);
int dispatcher_gpu(float *coord_l, float *gain_l , long size_x, long size_y, long slice_n, int type, void* source, long int src_size);

__global__ void mutiplier_kernel_type_c(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n);
__global__ void mutiplier_kernel_type_su(float *coord, float *gain, void* src,long size_x, long size_y, long slice_n);
__global__ void mutiplier_kernel_type_f(float *coord, float *gain, void* src,long size_x, long size_y, long slice_n);
void mutiplier_kernel_test(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n);

int main(int argc, char *argv[])
{
    char *raw_name,*out_name,*gain_name;
    FILE *file,*gain;
    //int threads;
    MrcHeader *inhead;

    raw_name=(char *)malloc(NAME*sizeof(char));
    out_name=(char *)malloc(NAME*sizeof(char));
    gain_name=(char *)malloc(NAME*sizeof(char));

    inhead=(MrcHeader *)malloc(sizeof(MrcHeader));

    if(argc!=4)
    {
        printf("Please input: raw_image gain_name out_name threads\n");
        return;
    }
    raw_name=argv[1];
    gain_name=argv[2];
    out_name=argv[3];
    //threads=atoi(argv[4]);

    file=fopen(raw_name,"rb");	
    mrc_read_head(file,inhead);
    fclose(file);

//    printf("raw_name %s, out_name %s, gain_name %s\n",raw_name,out_name,gain_name);
    defect_gain_correct(raw_name,gain_name,out_name,inhead,0);
	/*
	free(raw_name);
	free(out_name);
	free(gain_name);
	*/
	return 0;
}

int defect_gain_correct(char *fin, char *gain, char *fout, MrcHeader *head,int threads)
	{
	unsigned char lbuf[LINE];
	float fbuf[LINE];
	float *tmp_array,*gain_xy;
	//unsigned char *coor_xy;
	float *coor_xy;
	int i,j,x,y,p,k,w,w_row,w_length,m,range,point_s,point_e,n_file,size_bit,*pxy_array;
	long int n;
	FILE *input,*output,*gain_f, *temp_head;
	char *input_byte_c;short *input_byte_s;float *input_byte_f;short *input_byte_u;
	srand((unsigned) time(NULL));
	void *source =NULL;

	//open input output and gain file
	printf("Start defect and gain correction|\ninput %s output %s gain %s\n",fin,fout,gain);
	input=fopen(fin,"rb");
	gain_f=fopen(gain,"rb");

	//revise input image header then written for output file
	int size_x=head->nx;
	int size_y=head->ny;
	int slice_n=head->nz;
        int file_type=head->mode;
	head->mode=2;
        switch(file_type)
                {
                case 0:size_bit=1;break;
                case 1:size_bit=2;break;
                case 2:size_bit=4;break;
                case 6:size_bit=2;break;
                default:printf("File type error!");
                }

	//calculate the size of the input and gain file by byte.
	fseek(input,0,SEEK_END);
	long int input_size=ftell(input);
	rewind(input);

	fseek(gain_f,0,SEEK_END);
	long int gain_f_size=ftell(gain_f);
	rewind(gain_f);

	//skip header
	fseek(input,1024,0);
	fseek(gain_f,1024,0);
	//printf("input_size %d file_type %d size_bit %d\n",input_size,file_type,size_bit);

	//malloc memory for the file pointer
	float *gain_byte=(float*)malloc(sizeof(char*)*(gain_f_size-1024));

	//malloc memory for pxy_array
        pxy_array=(int*)malloc(sizeof(int*)*4000);

	//malloc memory for variate 
	tmp_array=(float*)malloc(sizeof(float)*size_x);

	gain_xy=(float*)malloc(sizeof(float)*size_y*size_x);
	//coor_xy=(unsigned char*)malloc(sizeof(unsigned char)*size_y*size_x*slice_n);
	coor_xy=(float*)malloc(sizeof(float)*size_y*size_x*slice_n);


	//read the input and gain file into memory
	//note: the byte size(second parameter) is basen on the type of the file

	switch(file_type)
                {
                case 0:size_bit=1;input_byte_c=(char*)malloc(sizeof(char*)*(input_size-1024));fread(input_byte_c,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_c;
				break;
                case 1:size_bit=2;input_byte_s=(short*)malloc(sizeof(char*)*(input_size-1024));fread(input_byte_s,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_s;
				break;
                case 2:size_bit=4;input_byte_f=(float*)malloc(sizeof(char*)*(input_size-1024));fread(input_byte_f,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_f;
				break;
                case 6:size_bit=2;input_byte_u=(short*)malloc(sizeof(char*)*(input_size-1024));fread(input_byte_u,size_bit,(input_size-1024)/size_bit,input);
				source = input_byte_u;
				break;
                default:printf("File type error!");
                }

	fread(gain_byte,sizeof(float),(gain_f_size-1024)/4,gain_f);

	//read gain 
	float badcut=6.0;
	long int num_gain=0;
	int defect_num=0;
	clock_t start,finish;

	long int total_num=0,gain_num=0,gain_m=0;
	
	start=clock();
	for(n=0; n<size_x*size_y; n++)
		{
		gain_xy[n]=(*(float*)(gain_byte+n))*10.0;
		}
	finish=clock();
	printf("read gain_time %d \n",finish-start);
	start=clock();
	//#pragma omp parallel for num_threads(threads) private(gain_num)
	//#pragma omp parallel for schedule(dynamic)
	/*
		for(n=0; n<size_x*size_y*slice_n; n++)
			{
			gain_num=n%(size_x*size_y);
                        switch(file_type)
                                {
                                case 0:coor_xy[n]=(*(unsigned char*)(input_byte_c+n));coor_xy[n]=(coor_xy[n])*(gain_xy[gain_num]);break;
                                case 1:coor_xy[n]=(*(short*)(input_byte_s+n));coor_xy[n]=(coor_xy[n])*(gain_xy[gain_num]);break;
                                case 2:coor_xy[n]=(*(float*)(input_byte_f+n));coor_xy[n]=(coor_xy[n])*(gain_xy[gain_num]);break;
                                case 6:coor_xy[n]=(*(short*)(input_byte_u+n));coor_xy[n]=(coor_xy[n])*(gain_xy[gain_num]);break;
                                default:printf("File type error!\n");
                                }
			
			}
	*/
	int indicator=0;
	indicator=dispatcher_gpu( coor_xy, gain_xy, size_x, size_y, slice_n, file_type, source, input_size-1024 );

	//defect correction for points
	finish=clock();
	printf("gain time %d s \n",(finish-start)/CLOCKS_PER_SEC);

	fclose(input);
	fclose(gain_f);
	free(gain_byte);
	switch(file_type)
               {
               case 0:free(input_byte_c);break;
               case 1:free(input_byte_s);break;
               case 2:free(input_byte_f);break;
               case 6:free(input_byte_u);break;
               default:printf("File type error!\n");
               }
	free(tmp_array);


	//write output file
	output=fopen(fout,"wb");
	//write head for output file
	fwrite(head,(sizeof(MrcHeader)+head->next),1,output);
	start=clock();
	//for(n=0;n<size_x*size_y*slice_n;n++)
		{
		//fwrite(&coor_xy[n],sizeof(unsigned char),1,output);
		//fwrite(&coor_xy[n],sizeof(float),1,output);
		fwrite(coor_xy,sizeof(float),size_x*size_y*slice_n,output);
		}
		printf("entrice written: %ld\n", size_x*size_y*slice_n);
		printf("head length: %d\n", (sizeof(MrcHeader)+head->next));
	finish=clock();
	printf("write_time %d \n",finish-start);

	start=clock();
	free(coor_xy);
	fclose(output);
	finish=clock();
	printf("free time %d \n",finish-start);
	printf("Defect and gain correction finished!\n");

	printf("En Taro Tassaddar!!!\n");
	return 0;
}

/**********************/

int dispatcher_gpu(float *coord_l, float *gain_l , long size_x, long size_y, long slice_n, int type, void* source, long int src_size){
	//set up cuda 
	hipSetDevice(1);	
	void *device_coord=NULL;
	void *device_gain=NULL;
	void *device_src=NULL;
	//void *temp_coord=NULL;
	//void *temp_gain=NULL;
	
	hipMalloc( (void **)&device_coord,  sizeof(float)*size_x*size_y*slice_n  );
	hipMalloc( (void **)&device_gain,  sizeof(float)*size_x*size_y );
	hipMalloc( (void **)&device_src, sizeof(char*)*src_size );
	//data transfer to device
	hipMemcpy( device_src, source, sizeof(char*)*src_size, hipMemcpyHostToDevice );
	hipMemcpy( device_gain, gain_l, sizeof(float)*size_x*size_y, hipMemcpyHostToDevice );

	//activate kerneld
	switch(type){
		case 0://c
			mutiplier_kernel_type_c<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, device_src, size_x, size_y, slice_n );
			//mutiplier_kernel_test( coord_l, gain_l, source, size_x, size_y, slice_n );
			break;
		case 1://s
			mutiplier_kernel_type_su<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, device_src, size_x, size_y, slice_n );			
			break;
		case 2://f
			mutiplier_kernel_type_f<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, device_src, size_x, size_y, slice_n );
			break;
		case 6://u
			mutiplier_kernel_type_su<<< GRID_BLOCK, BLOCK_SIZE >>>( (float*)device_coord, (float*)device_gain, device_src, size_x, size_y, slice_n );			
			break;
		default:
			printf("Well well, Houston, we have some problem....\n");
			exit(1);
	}
	//data transfer from device
	hipMemcpy( coord_l, device_coord, sizeof(float)*size_x*size_y*slice_n, hipMemcpyDeviceToHost );
	//hipMemcpy( coord_l, device_coord, size_x*size_y*slice_n, hipMemcpyDeviceToHost );
	//hipMemcpy( gain_l, device_gain, size_x*size_y,hipMemcpyDeviceToHost );
	
	//clean up
	hipFree( device_coord );
	hipFree( device_gain );

	return 117;
}

__global__ void mutiplier_kernel_type_c(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n){
	//execute all those mutiplications of type_c data
	long index=0; 
	//long unit_n = size_x*size_y*slice_n/GRID_BLOCK;
	long unit_n = UNIT_N;
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	for (int i=0; i<unit_n; i++){
		if (index*unit_n + i >= size_x*size_y*slice_n ){ //boundary check
			return;
		}
		slice_c = (index*unit_n+i)%(size_x*size_y);
		coord[index*unit_n+i] = (*(((unsigned char*)src)+index*unit_n+i))*gain[slice_c];
	}
	//__threadfence()
	return;
}
__global__ void mutiplier_kernel_type_su(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n){//execute all those mutiplications of type_s data
	//execute all those mutiplications of type_c data
	long index=0; 
	//long unit_n = size_x*size_y*slice_n/GRID_BLOCK;
	long unit_n = UNIT_N;
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	for (int i=0; i<unit_n; i++){
		if (index*unit_n + i >= size_x*size_y*slice_n ){ //boundary check
			return;
		}
		slice_c = (index*unit_n+i)%(size_x*size_y);
		coord[index*unit_n+i] = (*(((short*)src)+index*unit_n+i))*gain[slice_c];
	}
	//__threadfence()
	return;
}
__global__ void mutiplier_kernel_type_f(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n){//execute all those mutiplications of type_f data
	//execute all those mutiplications of type_c data
	long index=0; 
	//long unit_n = size_x*size_y*slice_n/GRID_BLOCK;
	long unit_n = UNIT_N;
	index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	for (int i=0; i<unit_n; i++){
		if (index*unit_n + i >= size_x*size_y*slice_n ){ //boundary check
			return;
		}
		slice_c = (index*unit_n+i)%(size_x*size_y);
		coord[index*unit_n+i] = (*(((float*)src)+index*unit_n+i))*gain[slice_c];
	}
	//__threadfence()
	return;
}

void mutiplier_kernel_test(float *coord, float *gain, void* src, long size_x, long size_y, long slice_n){
	///////////////////////////////
	printf("Activating kernel test\n");
	//execute all those mutiplications of type_c data
	long index=0; 
	//long unit_n = size_x*size_y*slice_n/GRID_BLOCK;
	//long unit_n = UNIT_N;
	long unit_n = size_x*size_y*slice_n;
	//index = blockDim.x*blockIdx.x + threadIdx.x;  //such fun
	long slice_c=0;

	for (int i=0; i<unit_n; i++){
		
		if (index*unit_n + i >= size_x*size_y*slice_n ){ //boundary check
			return;
		}
		
		slice_c = (index*unit_n+i)%(size_x*size_y);
		//slice_c = (i)%(size_x*size_y);
		coord[index*unit_n+i] = (*(((unsigned char*)src)+index*unit_n+i))*gain[slice_c];
		//coord[i] = (*(((unsigned char*)src)+i))*gain[slice_c];
	}
	//__threadfence()
	printf("Kernel test done\n");
	return;
}
